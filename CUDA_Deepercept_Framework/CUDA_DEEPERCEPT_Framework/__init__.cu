#include "hip/hip_runtime.h"
#include "perceptor.h"
#include "srcMeasure.h"

int main(void) {
	Perceptor p1(1);
	Perceptor p2(0);
	SrcMeasure sm;
	p1.getGpuInformation();
	p1.getCuBlasVersion();
	p1.getCuDnnVersion();
	p1.getGpuDriverVersion();
	p1.setSynchronizeGpuStream(false);
	p2.setSynchronizeGpuStream(false);
	
	Tensor m1({ 3, 3 }, "m1", 1.0);
	Tensor m2({ 3, 3 }, "m2", 1.0);

	m1[{ 9, 2, 3,
		4, 5, 6,
		7, 8, 9}];

	m2[{ 1, 0, 0,
		0, 1, 0,
		0, 0, 1}];

	m1.show();
	m2.show();

	Tensor* m3 = p1.matAdd(&m1, &m2);
	m3->setName("m1 + m2");
	p1.retrievDataFromDevice(m3);
	m3->show();

	Tensor* m4 = p1.matMult(&m1, &m2);
	m4->setName("m1 x m2");
	p1.retrievDataFromDevice(m4);
	m4->show();

	Tensor* m5 = p1.matSub(&m1, &m2);
	m5->setName("m1 - m2");
	p1.retrievDataFromDevice(m5);
	m5->show();

	p1.matMult(2.0, &m1);
	p1.retrievDataFromDevice(&m1);
	m1.show();

	p1.matSwap(&m1, &m2);
	p1.retrievDataFromDevice(&m1);
	p1.retrievDataFromDevice(&m2);
	m1.show();
	m2.show();

	int i = p1.matMaxIndex(&m1);
	int j = p1.matMaxIndex(&m2);
	cout << "m1 max : " << i << endl;
	cout << "m2 max : " << j << endl;

	i = p1.matMinIndex(&m1);
	j = p1.matMinIndex(&m2);
	cout << "m1 min : " << i << endl;
	cout << "m2 min : " << j << endl;

	dtype sum = p1.matSum(&m1);
	dtype sum2 = p1.matSum(&m2);
	cout << "m1 sum : " << sum << endl;
	cout << "m2 sum : " << sum2 << endl;

	Tensor m6({ 1, 3 }, 1.0);
	Tensor m7({ 3, 1 }, 2.0);

	m6.show();
	m7.show();

	Tensor* m8 = p1.matMult(&m6, &m7);
	p1.retrievDataFromDevice(m8);
	m8->show();

	int size1 = 4096;
	int size2 = 4096;

	Tensor m12({ 4096, 4096 });
	Tensor m9({ size1, size2 }, 1.0);
	for (int i = 0; i < size1; i++) 
		for (int j = 0; j < size2; j++)
			m9(i, j) = i * size2 + j;

	m9.show(9, 0);
	for (int i = 0; i < 10; i++) {
		sm.startTime(0);
		p1.matTranspose(&m9);
		p1.retrievDataFromDevice(&m9);
		sm.endTime(0, "Transpose time");
		m9.show(9, 0);
	}
	
	Tensor m10({ 4096, 4096 });
	Tensor m11({ 4096, 4096 });
	for (int i = 0; i < 4096; i++)
		for (int j = 0; j < 4096; j++) {
			m10(i, j) = float(i) / float(j + 1);
			m11(j, i) = float(j) / float(i + 1);
		}
	Tensor m13({ 4096, 4096 });
	Tensor m14({ 4096, 4096 });
	for (int i = 0; i < 4096; i++)
		for (int j = 0; j < 4096; j++) {
			m13(i, j) = float(i) / float(j + 1);
			m14(j, i) = float(j) / float(i + 1);
		}

	p1.sendToDevice(&m10);
	p1.sendToDevice(&m11);
	p2.sendToDevice(&m13);
	p2.sendToDevice(&m14);

	Tensor* temp = new Tensor({ 4096, 4096 });
	p1.sendToDevice(temp);
	Tensor* temp2 = new Tensor({ 4096, 4096 });
	p2.sendToDevice(temp2);

	sm.startTime(0);
	for (int i = 0; i < 200; i++) {
		p1.matMult(temp, &m10, &m11);
		p2.matMult(temp2, &m13, &m14);
	}
	p1.retrievDataFromDevice(temp);
	p2.retrievDataFromDevice(temp2);
	double elapsedTime = sm.endTime(0, "Alloc time 2");

	temp->show();
	temp2->show();
}