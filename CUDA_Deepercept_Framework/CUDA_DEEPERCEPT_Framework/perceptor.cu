#include "hip/hip_runtime.h"
#include "perceptor.h"
#include "srcMeasure.h"

void Perceptor::getGpuInformation() {
	int nDevices;
	hipGetDeviceCount(&nDevices);

	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);

		string archName = "";
		int cores = 0;
		int mp = prop.multiProcessorCount;
		switch (prop.major) {
		case 2: // Fermi
			if (prop.minor == 1) cores = mp * 48;
			else cores = mp * 32;
			archName = "Fermi";
			break;
		case 3: // Kepler
			cores = mp * 192;
			archName = "Kepler";
			break;
		case 5: // Maxwell
			cores = mp * 128;
			archName = "Maxwell";
			break;
		case 6: // Pascal
			if (prop.minor == 1) cores = mp * 128;
			else if (prop.minor == 0) cores = mp * 64;
			else printf("Unknown device type\n");
			archName = "Pascal";
			break;
		case 7: // Volta
			if (prop.minor == 0) cores = mp * 64;
			else printf("Unknown device type\n");
			archName = "Volta";
			break;
		default:
			printf("Unknown device type\n");
			break;
		}

		printf("Device Number : %d\n", i);
		printf("\tDevice Name : %s [%s]\n", prop.name, archName.c_str());
		printf("\tCompute Capability : %d.%d\n", prop.major, prop.minor);
		printf("\tGPU Clock Rate (GHz) : %f\n", float(prop.clockRate) / (1000.0 * 1000.0));
		printf("\tMemory Clock Rate (GHz) : %f\n", float(prop.memoryClockRate) / (1000.0 * 1000.0));
		printf("\tMemory Size (GB) : %f\n", static_cast<float>(prop.totalGlobalMem) / (1024 * 1024 * 1024));
		printf("\tMemory Bus Width (bits) : %d\n", prop.memoryBusWidth);
		printf("\tPeak Memory Bandwitdh (GB/s) : %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8.0) / 1.0e6);
		printf("\tNumber of Multi Processor : %d\n", prop.multiProcessorCount);
		printf("\tNumber of Cuda Cores : %d\n", cores);
		printf("\tMax Grid Size : [%d, %d, %d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\tMax Threads Dimension : [%d, %d, %d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("\tMax Threads Per Block : %d\n", prop.maxThreadsPerBlock);
		printf("\tMax Threads Per Multi Processor : %d\n", prop.maxThreadsPerMultiProcessor);
	}
}

void Perceptor::getCuDnnVersion() {
	cout << "cuDNN version : " << hipdnnGetVersion() << endl;
}

void Perceptor::getCuBlasVersion() {
	int version;
	cublasGetVersion(cuBlasHandle, &version);
	cout << "cuBlas version : " << version << endl;
}
void Perceptor::getGpuDriverVersion() {
	int version;
	hipDriverGetVersion(&version);
	cout << "GPU driver version : " << version << endl;
}

// Input pointer of tensor A and B
// Output is tensor pointer of result
Tensor* Perceptor::matSgemm(Tensor* tA, Tensor* tB, float alpha, float beta) {
	if (tA->shape()[1] != tB->shape()[0]) {
		cout << "Cannot multiply " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->row(), tB->col() }, false);
	t_out->sendToDevice();

	// Allocate to device
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With row major!
	CuBLAS_ERROR(
		hipblasSgemm(
			cuBlasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			tB->col(), tA->row(), tA->col(), // Rows of OP(A), Columns of OP(B), Rows of C
			&alpha, // alpha
			tB->devDataPtr(), tB->col(), // A, leading dimension of A used to store the matrix A
			tA->devDataPtr(), tA->col(), // B, leading dimension of B used to store the matrix B
			&beta, // beta
			t_out->devDataPtr(), tB->col() // C, leading dimension of C
		)
	);
	syncGpuStream();

	return t_out;
}

void Perceptor::matSgemm(Tensor* tOut, Tensor* tA, Tensor* tB, float alpha, float beta) {
	if (tA->shape()[1] != tB->shape()[0]) {
		cout << "Cannot multiply " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate to device
	if (!tOut->haveDevicePtr())
		tOut->sendToDevice();
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With row major!
	CuBLAS_ERROR(
		hipblasSgemm(
			cuBlasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			tB->col(), tA->row(), tA->col(), // Rows of OP(A), Columns of OP(B), Rows of C
			&alpha, // alpha
			tB->devDataPtr(), tB->col(), // A, leading dimension of A used to store the matrix A
			tA->devDataPtr(), tA->col(), // B, leading dimension of B used to store the matrix B
			&beta, // beta
			tOut->devDataPtr(), tB->col() // C, leading dimension of C
		)
	);
	syncGpuStream();
}

Tensor* Perceptor::matMult(Tensor* tA, Tensor* tB) {
	return matSgemm(tA, tB, 1, 0);
}

void Perceptor::matMult(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgemm(tOut, tA, tB, 1, 0);
}

void Perceptor::matMult(dtype scalA, Tensor* tB) {
	// Allocate result tensor
	Tensor* t_out = new Tensor({ tB->row(), tB->col() }, false);
	t_out->sendToDevice();

	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	CuBLAS_ERROR(
		hipblasSscal(
			cuBlasHandle,
			tB->size(),
			&scalA,
			tB->devDataPtr(),
			1
		)
	);

	syncGpuStream();
}

Tensor* Perceptor::matSgeam(Tensor* tA, Tensor* tB, float alpha, float beta) {
	if (!tA->isSame(*tB)) {
		cout << "Cannot sum " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}
	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->row(), tB->col() }, false);
	t_out->sendToDevice();

	// Allocate to device
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	CuBLAS_ERROR(
		hipblasSgeam(
			cuBlasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			tB->col(), tA->row(),
			&alpha,
			tB->devDataPtr(), tB->col(),
			&beta,
			tA->devDataPtr(), tA->col(),
			t_out->devDataPtr(), tB->col()
		)
	);
	syncGpuStream();

	return t_out;
}

void Perceptor::matSgeam(Tensor* tOut, Tensor* tA, Tensor* tB, float alpha, float beta) {
	if (!tA->isSame(*tB)) {
		cout << "Cannot sum " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}
	// Allocate to device
	if (!tOut->haveDevicePtr())
		tOut->sendToDevice();
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With row major!
	CuBLAS_ERROR(
		hipblasSgeam(
			cuBlasHandle,
			HIPBLAS_OP_N, HIPBLAS_OP_N,
			tB->col(), tA->row(),
			&alpha,
			tB->devDataPtr(), tB->col(),
			&beta,
			tA->devDataPtr(), tA->col(),
			tOut->devDataPtr(), tB->col()
		)
	);
	syncGpuStream();
}

Tensor* Perceptor::matAdd(Tensor* tA, Tensor* tB) {
	return matSgeam(tA, tB, 1, 1);
}

void Perceptor::matAdd(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgeam(tOut, tA, tB, 1, 1);
}

Tensor* Perceptor::matSub(Tensor* tA, Tensor* tB) {
	return matSgeam(tA, tB, -1, 1);
}

void Perceptor::matSub(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgeam(tOut, tA, tB, -1, 1);
}

__global__ void cuEltwiseMultiplication(dtype* tOut, dtype* tA, dtype* tB, int row, int col) {
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;

	if (y < row && x < col)
		tOut[y * col + x] = tA[y * col + x] * tB[y * col + x];
}

// Return = tA * tB (Haramard Product)
Tensor* Perceptor::matEltMult(Tensor* tA, Tensor* tB) {
	if (!tA->isSame(*tB)) {
		cout << "Cannot element wise multiplication with " << tA->name() << " and " << tB->name() << ", their shape is different" << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->row(), tA->col() }, false);
	t_out->sendToDevice();

	// Allocate to device
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	cuEltwiseMultiplication <<<blocks, threads >>> (t_out->devDataPtr(), tA->devDataPtr(), tB->devDataPtr(), tA->row(), tA->col());
	syncGpuStream();
}

// tOut = tA * tB (Haramard Product)
void Perceptor::matEltMult(Tensor* tOut, Tensor* tA, Tensor* tB) {
	if (!tA->isSame(*tB)) {
		cout << "Cannot element wise multiplication with " << tA->name() << " and " << tB->name() << ", their shape is different" << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate to device
	if (!tOut->haveDevicePtr())
		tOut->sendToDevice();
	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	cuEltwiseMultiplication << <blocks, threads >> > (tOut->devDataPtr(), tA->devDataPtr(), tB->devDataPtr(), tA->row(), tA->col());
	syncGpuStream();
}

void Perceptor::matSwap(Tensor* tA, Tensor* tB, bool forceSwap) {
	if (!forceSwap && !tA->isSame(*tB)) {
		cout << tA->name() << " and " << tB->name() << " tensors shapes are different!" << endl;
		exit(EXIT_FAILURE);
	}
	if(forceSwap && tA->size() != tB->size()) {
		cout << tA->name() << " and " << tB->name() << " tensors size are different!" << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	CuBLAS_ERROR(
		hipblasSswap(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			tB->devDataPtr(),
			1
		)
	);
	syncGpuStream();
}

void Perceptor::matCopy(Tensor* tB, Tensor* tA) {
	if (!tA->isSame(*tB)) {
		cout << tA->name() << " and " << tB->name() << " tensors shapes are different!" << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		tA->sendToDevice();
	if (!tB->haveDevicePtr())
		tB->sendToDevice();

	CuBLAS_ERROR(
		hipblasScopy(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			tB->devDataPtr(),
			1
		)
	);
	syncGpuStream();
}

int Perceptor::matMaxIndex(Tensor* tA) {
	int result = 0;

	if (!tA->haveDevicePtr())
		tA->sendToDevice();

	CuBLAS_ERROR(
		hipblasIsamax(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();
	
	return result - 1;
}

int Perceptor::matMinIndex(Tensor* tA) {
	int result = 0;

	if (!tA->haveDevicePtr())
		tA->sendToDevice();

	CuBLAS_ERROR(
		hipblasIsamin(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();

	return result - 1;
}

dtype Perceptor::matSum(Tensor* tA) {
	dtype result = 0;

	if (!tA->haveDevicePtr())
		tA->sendToDevice();

	CuBLAS_ERROR(
		hipblasSasum(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();

	return result;
}

__global__ void cuMatrixCopy(const dtype* src, dtype* dst, int src_row, int src_col, int dst_row, int dst_col) {
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	if (y < dst_row && x < dst_col && y < src_row && x < src_col)
		dst[y * dst_col + x] = src[y * src_col + x];
}

__global__ void iptransposeCoalesced(dtype* src_data, dtype* dummy_data, int src_row, int src_col, int dummy_row, int dummy_col)
{
	__shared__ float tile_s[BLOCK_DIM][BLOCK_DIM + 1];
	__shared__ float tile_d[BLOCK_DIM][BLOCK_DIM + 1];

	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if (blockIdx.y > blockIdx.x) { // handle off-diagonal case
		int dx = blockIdx.y * BLOCK_DIM + threadIdx.x;
		int dy = blockIdx.x * BLOCK_DIM + threadIdx.y;
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if(y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) * src_col + x];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				tile_d[threadIdx.y + j][threadIdx.x] = src_data[(dy + j) * src_col + dx];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(dy + j) * dummy_col + dx] = tile_s[threadIdx.x][threadIdx.y + j];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(y + j) * dummy_col + x] = tile_d[threadIdx.x][threadIdx.y + j];
	}

	else if (blockIdx.y == blockIdx.x) { // handle on-diagonal case
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j)*src_col + x];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(y + j) * dummy_col + x] = tile_s[threadIdx.x][threadIdx.y + j];
	}
}

__global__ void iptransposeCoalesced(dtype* src_data, int src_row, int src_col)
{
	__shared__ float tile_s[BLOCK_DIM][BLOCK_DIM + 1];
	__shared__ float tile_d[BLOCK_DIM][BLOCK_DIM + 1];

	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if (blockIdx.y > blockIdx.x) { // handle off-diagonal case
		int dx = blockIdx.y * BLOCK_DIM + threadIdx.x;
		int dy = blockIdx.x * BLOCK_DIM + threadIdx.y;
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) * src_col + x];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				tile_d[threadIdx.y + j][threadIdx.x] = src_data[(dy + j) * src_col + dx];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				src_data[(dy + j) * src_col + dx] = tile_s[threadIdx.x][threadIdx.y + j];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				src_data[(y + j) * src_col + x] = tile_d[threadIdx.x][threadIdx.y + j];
	}

	else if (blockIdx.y == blockIdx.x) { // handle on-diagonal case
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j)*src_col + x];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				src_data[(y + j) * src_col + x] = tile_s[threadIdx.x][threadIdx.y + j];
	}
}

void Perceptor::matTranspose(Tensor* tA) {
	if (tA->dimension() != 2) {
		cout << "Cannot transpose matrix. " << tA->name() << " is " << tA->dimension() << " dimension matrix." << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		tA->sendToDevice();

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, VECTOR_SIZE);
	dim3 threads2(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	if (tA->shape(0) == tA->shape(1)) {
		iptransposeCoalesced <<< blocks, threads >>> (tA->devDataPtr(), tA->col(), tA->row());
		syncGpuStream();
	}
	else {
		iptransposeCoalesced <<< blocks, threads >>>
			(tA->devDataPtr(), dummyTensor->devDataPtr(), tA->row(), tA->col(), MATRIX_DIM_LIMIT, MATRIX_DIM_LIMIT);
		syncGpuStream();

		cuMatrixCopy <<< blocks, threads2 >>>
			(dummyTensor->devDataPtr(), tA->devDataPtr(), MATRIX_DIM_LIMIT, MATRIX_DIM_LIMIT, tA->col(), tA->row());
		syncGpuStream();
	}

	tA->swapDimension(0, 1);
}

