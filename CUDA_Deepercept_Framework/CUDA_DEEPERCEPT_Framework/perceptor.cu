#include "hip/hip_runtime.h"
#include "perceptor.h"
#include "srcMeasure.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

bool Perceptor::gpuUtilization[] = { 0 };

void Perceptor::getGpuInformation() {
	int nDevices;
	hipGetDeviceCount(&nDevices);

	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);

		string archName = "";
		int cores = 0;
		int mp = prop.multiProcessorCount;
		switch (prop.major) {
		case 2: // Fermi
			if (prop.minor == 1) cores = mp * 48;
			else cores = mp * 32;
			archName = "Fermi";
			break;
		case 3: // Kepler
			cores = mp * 192;
			archName = "Kepler";
			break;
		case 5: // Maxwell
			cores = mp * 128;
			archName = "Maxwell";
			break;
		case 6: // Pascal
			if (prop.minor == 1) cores = mp * 128;
			else if (prop.minor == 0) cores = mp * 64;
			else printf("Unknown device type\n");
			archName = "Pascal";
			break;
		case 7: // Volta
			if (prop.minor == 0) cores = mp * 64;
			else printf("Unknown device type\n");
			archName = "Volta";
			break;
		default:
			printf("Unknown device type\n");
			break;
		}

		size_t total_size, free_size;

		printf("Device Number : %d\n", i);
		printf("\tDevice Name : %s [%s]\n", prop.name, archName.c_str());
		printf("\tCompute Capability : %d.%d\n", prop.major, prop.minor);
		printf("\tGPU Clock Rate (GHz) : %.2f\n", float(prop.clockRate) / (1000.0 * 1000.0));
		printf("\tMemory Clock Rate (GHz) : %.2f\n", float(prop.memoryClockRate) / (1000.0 * 1000.0));
		printf("\tMemory Size (GB) : %.2f\n", double(static_cast<long long>(prop.totalGlobalMem)) / (1024 * 1024 * 1024));
		setDevice(i); hipMemGetInfo(&free_size, &total_size);
		printf("\tFree Memory Size (GB) : %.2f\n", double(static_cast<long long>(free_size)) / (1024 * 1024 * 1024));
		printf("\tMemory Bus Width (bits) : %d\n", prop.memoryBusWidth);
		printf("\tPeak Memory Bandwitdh (GB/s) : %.2f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8.0) / 1.0e6);
		printf("\tShared Memory per Blocks (KB) : %.2f\n", double(static_cast<long long>(prop.sharedMemPerBlock)) / 1024);
		printf("\tNumber of Multi Processor : %d\n", prop.multiProcessorCount);
		printf("\tNumber of Cuda Cores : %d\n", cores);
		printf("\tMax Grid Size : [%d, %d, %d]\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\tMax Threads Dimension : [%d, %d, %d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("\tMax Threads Per Block : %d\n", prop.maxThreadsPerBlock);
		printf("\tMax Threads Per Multi Processor : %d\n", prop.maxThreadsPerMultiProcessor);
	}
	setDevice();
}

void Perceptor::getCuDnnVersion() {
	cout << "cuDNN version : " << hipdnnGetVersion() << endl;
}

void Perceptor::getCuBlasVersion() {
	int version;
	cublasGetVersion(cuBlasHandle, &version);
	cout << "cuBlas version : " << version << endl;
}
void Perceptor::getGpuDriverVersion() {
	int version;
	hipDriverGetVersion(&version);
	cout << "GPU driver version : " << version << endl;
}

void Perceptor::checkDevice(Tensor* tA, Tensor* tB) {
	checkDevice(tA);
	checkDevice(tB);
	if (tA->deviceId() != tB->deviceId()) {
		cout << "Device of " << tA->name() << " = " << tA->deviceId() << 
			" and device of " << tB->name() << " = " << tB->deviceId() << " are different" << endl;
		exit(EXIT_FAILURE);
	}
	setDevice();
}

void Perceptor::checkDevice(Tensor* tA) {
	if (!tA->haveDevicePtr() && !tA->haveDeviceDataPtr()) {
		if (tA->deviceId() != deviceId()) {
			cout << "Device of " << tA->name() << " = " << tA->deviceId() << " is automatically chanaged to " << deviceId()
				 << " because " << tA->name() << " was not allocated at device" << endl;
		}
		tA->setDevice(deviceId());
		setDevice();
	}
	else if (deviceId() != tA->deviceId()) {
		cout << "Device of Perceptor = " << deviceId() <<
			" and device of " << tA->name() << " = " << tA->deviceId() << " are different" << endl;
		exit(EXIT_FAILURE);
	}
}

// Input pointer of tensor A and B
// Output is tensor pointer of result
Tensor* Perceptor::matSgemm(Tensor* tA, Tensor* tB, float alpha, float beta) {
	checkDevice(tA, tB);
	if (tA->col() != tB->row()) {
		cout << "Cannot multiply " << tA->name() << " and " << tB->name() << endl;
		cout << "Number of " << tA->name() << " columns and number of " << tB->name() << " rows are different" << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->col(), tB->row() }, false);
	t_out->setDevice(deviceId());
	sendToDevice(t_out);

	// Allocate to device
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With column major!
	CuBLAS_ERROR(
		hipblasSgemm(
			cuBlasHandle, // Handle
			HIPBLAS_OP_N, HIPBLAS_OP_N, // Trans A, Trans B
			tA->row(), tB->col(), tA->col(), // Rows of OP(A), Columns of OP(B), Rows of C
			&alpha, // alpha
			tA->devDataPtr(), tA->row(), // A, leading dimension of A used to store the matrix A
			tB->devDataPtr(), tB->row(), // B, leading dimension of B used to store the matrix B
			&beta, // beta
			t_out->devDataPtr(), tA->row() // C, leading dimension of C
		)
	);
	syncGpuStream();

	return t_out;
}

void Perceptor::matSgemm(Tensor* tOut, Tensor* tA, Tensor* tB, float alpha, float beta) {
	checkDevice(tOut, tA);
	checkDevice(tA, tB);
	if (tA->shape()[1] != tB->shape()[0]) {
		cout << "Cannot multiply " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate to device
	if (!tOut->haveDevicePtr())
		sendToDevice(tOut);
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With row major!
	CuBLAS_ERROR(
		hipblasSgemm(
			cuBlasHandle, // Handle
			HIPBLAS_OP_N, HIPBLAS_OP_N, // Trans A, Trans B
			tA->row(), tB->col(), tA->col(), // Rows of OP(A), Columns of OP(B), Rows of C
			&alpha, // alpha
			tA->devDataPtr(), tA->row(), // A, leading dimension of A used to store the matrix A
			tB->devDataPtr(), tB->row(), // B, leading dimension of B used to store the matrix B
			&beta, // beta
			tOut->devDataPtr(), tA->row() // C, leading dimension of C
		)
	);
	syncGpuStream();
}

Tensor* Perceptor::matMult(Tensor* tA, Tensor* tB) {
	return matSgemm(tA, tB, 1, 0);
}

void Perceptor::matMult(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgemm(tOut, tA, tB, 1, 0);
}

void Perceptor::matMult(dtype scalA, Tensor* tB) {
	checkDevice(tB);
	// Allocate result tensor
	Tensor* t_out = new Tensor({ tB->col(), tB->row() }, false);
	t_out->setDevice(deviceId());
	sendToDevice(t_out);

	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	CuBLAS_ERROR(
		hipblasSscal(
			cuBlasHandle,
			tB->size(),
			&scalA,
			tB->devDataPtr(),
			1
		)
	);

	syncGpuStream();
}

Tensor* Perceptor::matSgeam(Tensor* tA, Tensor* tB, float alpha, float beta) {
	checkDevice(tA, tB);
	if (!tA->isSame(*tB)) {
		cout << "Cannot sum " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->col(), tA->row() }, false);
	t_out->setDevice(deviceId());
	sendToDevice(t_out);

	// Allocate to device
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	CuBLAS_ERROR(
		hipblasSgeam(
			cuBlasHandle, // Handle
			HIPBLAS_OP_N, HIPBLAS_OP_N, // Trans A, Trans B
			tA->col(), tA->row(), // m, n
			&alpha, // Alpha
			tA->devDataPtr(), tA->col(), // float *A, lda
			&beta, // Beta
			tB->devDataPtr(), tB->col(), // float *B, ldb
			t_out->devDataPtr(), tA->col() // float *C, ldc
		)
	);
	syncGpuStream();

	return t_out;
}

void Perceptor::matSgeam(Tensor* tOut, Tensor* tA, Tensor* tB, float alpha, float beta) {
	checkDevice(tOut, tA);
	checkDevice(tA, tB);
	if (!tA->isSame(*tB)) {
		cout << "Cannot sum " << tA->name() << " and " << tB->name() << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate to device
	if (!tOut->haveDevicePtr())
		sendToDevice(tOut);
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	// hipblasSgemm : C = alpha * (OP(A) * OP(B)) + beta * C
	// With row major!
	CuBLAS_ERROR(
		hipblasSgeam(
			cuBlasHandle, // Handle
			HIPBLAS_OP_N, HIPBLAS_OP_N, // Trans A, Trans B
			tA->col(), tA->row(), // m, n
			&alpha, // Alpha
			tA->devDataPtr(), tA->col(), // float *A, lda
			&beta, // Beta
			tB->devDataPtr(), tB->col(), // float *B, ldb
			tOut->devDataPtr(), tA->col() // float *C, ldc
		)
	);
	syncGpuStream();
}

Tensor* Perceptor::matAdd(Tensor* tA, Tensor* tB) {
	return matSgeam(tA, tB, 1, 1);
}

void Perceptor::matAdd(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgeam(tOut, tA, tB, 1, 1);
}

Tensor* Perceptor::matSub(Tensor* tA, Tensor* tB) {
	return matSgeam(tA, tB, -1, 1);
}

void Perceptor::matSub(Tensor* tOut, Tensor* tA, Tensor* tB) {
	matSgeam(tOut, tA, tB, -1, 1);
}

__global__ void cuEltwiseMultiplication(dtype* tOut, dtype* tA, dtype* tB, int row, int col) {
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;

	if (y < row && x < col)
		tOut[y + x * row] = tA[y + x * row] * tB[y + x * row];
}

// Return = tA * tB (Haramard Product)
Tensor* Perceptor::matEltMult(Tensor* tA, Tensor* tB) {
	checkDevice(tA, tB);
	if (!tA->isSame(*tB)) {
		cout << "Cannot element wise multiplication with " << tA->name() << " and " << tB->name() << ", their shape is different" << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate result tensor
	Tensor* t_out = new Tensor({ tA->col(), tA->row() }, false);
	sendToDevice(t_out);

	// Allocate to device
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	cuEltwiseMultiplication <<<blocks, threads >>> (t_out->devDataPtr(), tA->devDataPtr(), tB->devDataPtr(), tA->row(), tA->col());
	syncGpuStream();
}

// tOut = tA * tB (Haramard Product)
void Perceptor::matEltMult(Tensor* tOut, Tensor* tA, Tensor* tB) {
	checkDevice(tOut, tA);
	checkDevice(tA, tB);
	if (!tA->isSame(*tB)) {
		cout << "Cannot element wise multiplication with " << tA->name() << " and " << tB->name() << ", their shape is different" << endl;
		exit(EXIT_FAILURE);
	}

	// Allocate to device
	if (!tOut->haveDevicePtr())
		sendToDevice(tOut);
	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	cuEltwiseMultiplication << <blocks, threads >> > (tOut->devDataPtr(), tA->devDataPtr(), tB->devDataPtr(), tA->row(), tA->col());
	syncGpuStream();
}

void Perceptor::matSwap(Tensor* tA, Tensor* tB, bool forceSwap) {
	checkDevice(tA, tB);
	if (!forceSwap && !tA->isSame(*tB)) {
		cout << tA->name() << " and " << tB->name() << " tensors shapes are different!" << endl;
		exit(EXIT_FAILURE);
	}
	if(forceSwap && tA->size() != tB->size()) {
		cout << tA->name() << " and " << tB->name() << " tensors size are different!" << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	CuBLAS_ERROR(
		hipblasSswap(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			tB->devDataPtr(),
			1
		)
	);
	syncGpuStream();
}

void Perceptor::matCopy(Tensor* tB, Tensor* tA) {
	checkDevice(tB, tA);
	if (!tA->isSame(*tB)) {
		cout << tA->name() << " and " << tB->name() << " tensors shapes are different!" << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		sendToDevice(tA);
	if (!tB->haveDevicePtr())
		sendToDevice(tB);

	CuBLAS_ERROR(
		hipblasScopy(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			tB->devDataPtr(),
			1
		)
	);
	syncGpuStream();
}

int Perceptor::matMaxIndex(Tensor* tA) {
	checkDevice(tA);
	int result = 0;

	if (!tA->haveDevicePtr())
		sendToDevice(tA);

	CuBLAS_ERROR(
		hipblasIsamax(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();
	
	return result - 1;
}

int Perceptor::matMinIndex(Tensor* tA) {
	checkDevice(tA);
	int result = 0;

	if (!tA->haveDevicePtr())
		sendToDevice(tA);

	CuBLAS_ERROR(
		hipblasIsamin(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();

	return result - 1;
}

dtype Perceptor::matSum(Tensor* tA) {
	checkDevice(tA);
	dtype result = 0;

	if (!tA->haveDevicePtr())
		sendToDevice(tA);

	CuBLAS_ERROR(
		hipblasSasum(
			cuBlasHandle,
			tA->size(),
			tA->devDataPtr(),
			1,
			&result
		)
	);
	syncGpuStream();

	return result;
}

__global__ void cuMatrixCopy(const dtype* src, dtype* dst, int src_row, int src_col, int dst_row, int dst_col) {
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	if (y < dst_row && x < dst_col && y < src_row && x < src_col)
		dst[y + x * dst_row] = src[y + x * src_row];
}

__global__ void iptransposeCoalesced(dtype* src_data, dtype* dummy_data, int src_row, int src_col, int dummy_row, int dummy_col)
{
	__shared__ float tile_s[BLOCK_DIM][BLOCK_DIM + 1];
	__shared__ float tile_d[BLOCK_DIM][BLOCK_DIM + 1];

	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if (blockIdx.y > blockIdx.x) { // handle off-diagonal case
		int dx = blockIdx.y * BLOCK_DIM + threadIdx.x;
		int dy = blockIdx.x * BLOCK_DIM + threadIdx.y;
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if(y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) + x * src_row];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				tile_d[threadIdx.y + j][threadIdx.x] = src_data[(dy + j) + dx * src_row];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(dy + j) + dx * dummy_row] = tile_s[threadIdx.x][threadIdx.y + j];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(y + j) + x * dummy_row] = tile_d[threadIdx.x][threadIdx.y + j];
	}

	else if (blockIdx.y == blockIdx.x) { // handle on-diagonal case
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) + x * src_row];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			dummy_data[(y + j) + x * dummy_row] = tile_s[threadIdx.x][threadIdx.y + j];
	}
}

__global__ void iptransposeCoalesced(dtype* src_data, int src_row, int src_col)
{
	__shared__ float tile_s[BLOCK_DIM][BLOCK_DIM + 1];
	__shared__ float tile_d[BLOCK_DIM][BLOCK_DIM + 1];

	int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
	int y = blockIdx.y * BLOCK_DIM + threadIdx.y;

	if (blockIdx.y > blockIdx.x) { // handle off-diagonal case
		int dx = blockIdx.y * BLOCK_DIM + threadIdx.x;
		int dy = blockIdx.x * BLOCK_DIM + threadIdx.y;
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) + x * src_row];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				tile_d[threadIdx.y + j][threadIdx.x] = src_data[(dy + j) + dx * src_row];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (dy + j < src_row && dx < src_col)
				src_data[(dy + j) + dx * src_row] = tile_s[threadIdx.x][threadIdx.y + j];
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				src_data[(y + j) + x * src_row] = tile_d[threadIdx.x][threadIdx.y + j];
	}

	else if (blockIdx.y == blockIdx.x) { // handle on-diagonal case
		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				tile_s[threadIdx.y + j][threadIdx.x] = src_data[(y + j) + x * src_row];

		__syncthreads();

		for (int j = 0; j < BLOCK_DIM; j += VECTOR_SIZE)
			if (y + j < src_row && x < src_col)
				src_data[(y + j) + x * src_row] = tile_s[threadIdx.x][threadIdx.y + j];
	}
}

void Perceptor::matTranspose(Tensor* tA) {
	checkDevice(tA);
	if (tA->dimension() != 2) {
		cout << "Cannot transpose matrix. " << tA->name() << " is " << tA->dimension() << " dimension matrix." << endl;
		exit(EXIT_FAILURE);
	}

	if (!tA->haveDevicePtr())
		sendToDevice(tA);

	int nBblocks;
	if (tA->row() >= tA->col())
		nBblocks = tA->row() % BLOCK_DIM == 0 ? tA->row() / BLOCK_DIM : tA->row() / BLOCK_DIM + 1;
	else
		nBblocks = tA->col() % BLOCK_DIM == 0 ? tA->col() / BLOCK_DIM : tA->col() / BLOCK_DIM + 1;

	dim3 threads(BLOCK_DIM, VECTOR_SIZE);
	dim3 threads2(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks(nBblocks, nBblocks);

	if (tA->shape(0) == tA->shape(1)) {
		iptransposeCoalesced <<< blocks, threads >>> (tA->devDataPtr(), tA->col(), tA->row());
	}
	else {
		iptransposeCoalesced <<< blocks, threads >>>
			(tA->devDataPtr(), dummyTensor->devDataPtr(), tA->row(), tA->col(), MATRIX_DIM_LIMIT, MATRIX_DIM_LIMIT);
		syncGpuStream();

		cuMatrixCopy <<< blocks, threads2 >>>
			(dummyTensor->devDataPtr(), tA->devDataPtr(), MATRIX_DIM_LIMIT, MATRIX_DIM_LIMIT, tA->col(), tA->row());
	}

	syncGpuStream();
	tA->swapDimension(0, 1);
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init_rand(unsigned int seed, hiprandState_t* states) {

	/* we have to initialize the state */
	hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
		blockIdx.x, /* the sequence number should be different for each core (unless you want all
					cores to get the same sequence of numbers for some reason - use thread id! */
		0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
		&states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, dtype* data, dtype min, dtype max) {
	/* hiprand works like rand - except that it takes a state as a parameter */
	
	data[blockIdx.x] = (dtype(hiprand(&states[blockIdx.x])) / dtype(UINT32_MAX)) * (max - min) + min;
}


void Perceptor::matRand(Tensor* tA, dtype min, dtype max) {
	checkDevice(tA);
	if (!tA->haveDevicePtr())
		sendToDevice(tA);

	hiprandState_t* states;
	hipMalloc((void**)&states, tA->size() * sizeof(hiprandState_t));
	init_rand << <tA->size(), 1 >> > (time(0), states);
	randoms << <tA->size(), 1 >> > (states, tA->devDataPtr(), min, max);

	hipFree(states);
	syncGpuStream();
}


void Perceptor::sendToDevice(Tensor* t, bool sendData) {
	Tensor* devPtr = 0;
	t->setDevice(deviceId());
	hipSetDevice(deviceId());
	CUDA_CHECK(hipMalloc((void**)&devPtr, sizeof(Tensor)));
	CUDA_CHECK(hipMemcpy(devPtr, t, sizeof(Tensor), hipMemcpyHostToDevice));

	int* hostShape;
	CUDA_CHECK(hipMalloc((void**)&hostShape, sizeof(int) * t->dimension()));
	CUDA_CHECK(hipMemcpy(hostShape, t->mShape, sizeof(int) * t->dimension(), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(&(devPtr->mShape), &hostShape, sizeof(int*), hipMemcpyHostToDevice));

	int* hostCumulatedDimension;
	CUDA_CHECK(hipMalloc((void**)&hostCumulatedDimension, sizeof(int) * t->dimension()));
	CUDA_CHECK(hipMemcpy(hostCumulatedDimension, t->cumulatedDimension, sizeof(int) * t->dimension(), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(&(devPtr->cumulatedDimension), &hostCumulatedDimension, sizeof(int*), hipMemcpyHostToDevice));

	// Set device tensor as container
	CUDA_CHECK(hipMemcpy(&devPtr->isContainer, new bool(true), sizeof(bool), hipMemcpyHostToDevice));

	// Copy host data to device
	dtype* hostData;
	CUDA_CHECK(hipMalloc((void**)&hostData, sizeof(dtype) * t->size()));
	if (sendData)
		CUDA_CHECK(hipMemcpy(hostData, t->data, sizeof(dtype) * t->size(), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(&(devPtr->data), &hostData, sizeof(dtype*), hipMemcpyHostToDevice));
	t->devData = hostData;
	t->dev = devPtr;
	t->devShape = hostShape;
	t->devCumulatedDimension = hostCumulatedDimension;
	t->mHaveDevPtr = true;
	t->mHaveDevDataPtr = true;
}

void Perceptor::sendDataToDevice(Tensor* t) {
	if (t->deviceId() != deviceId()) {
		cout << t->name() << " and  perceptor device ID " << deviceId() << " are different" << endl;
		exit(EXIT_FAILURE);
	}
	if (t->haveDevicePtr()) {
		dtype* devPtr = 0;
		t->setDevice(deviceId());
		hipSetDevice(deviceId());
		hipFree(t->devData);
		CUDA_CHECK(hipMalloc((void**)&devPtr, sizeof(dtype) * t->size()));
		CUDA_CHECK(hipMemcpy(devPtr, t->data, sizeof(dtype) * t->size(), hipMemcpyHostToDevice));
		t->devData = devPtr;
		t->mHaveDevDataPtr = true;
	}
	else {
		cout << t->name() << " is not allocated at device" << endl;
		exit(EXIT_FAILURE);
	}
}

void Perceptor::retrievDataFromDevice(Tensor* t, bool retreiveOnlyData) {
	if (t->haveDevicePtr() && t->haveDeviceDataPtr()) {
		hipSetDevice(deviceId());
		CUDA_CHECK(hipMemcpy(t->data, t->devData, t->size() * sizeof(dtype), hipMemcpyDeviceToHost));
		if (!retreiveOnlyData) {
			Tensor temp;
			CUDA_CHECK(hipMemcpy(&temp, t->dev, sizeof(Tensor), hipMemcpyDeviceToHost));
			CUDA_CHECK(hipMemcpy(t->cumulatedDimension, temp.cumulatedDimension, t->dimension() * sizeof(int), hipMemcpyDeviceToHost));
			CUDA_CHECK(hipMemcpy(t->mShape, temp.mShape, t->dimension() * sizeof(int), hipMemcpyDeviceToHost));
			temp.setName("");
		}
	}
}

Perceptor* Perceptor::convolution(Tensor* tInput, Tensor* tFilter, Tensor* tOutput) {


	return this;
}